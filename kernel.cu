
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 512
#define NUM_BLOCK 16
#define MAX_SIZE 4096

__global__ void histo_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins)
{
	
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    __shared__ unsigned int histo_private[MAX_SIZE];
    
    if (threadIdx.x < MAX_SIZE) histo_private[threadIdx.x] = 0;
    __syncthreads();
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    int x = 0;
    while(x*stride+i < num_elements){
        atomicAdd(&histo_private[input[x*stride+i]], 1);
        ++x;
    }
    __syncthreads();
    
    //if (threadIdx.x < 7) {
      //  atomicAdd(&(histo[threadIdx.x]), private_histo[threadIdx.x] );
    //}
    x = 0;
    while(x*blockDim.x+threadIdx.x < num_bins){
        atomicAdd(&bins[x*blockDim.x+threadIdx.x], histo_private[x*blockDim.x+threadIdx.x]);
        ++x;
    }
	
	
	  /*************************************************************************/
}

void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

	  /*************************************************************************/
    //INSERT CODE HERE
    //dim3 DimGrid((n-1)/BLOCK_SIZE+1, (m-1)/BLOCK_SIZE+1, 1);
    //dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    histo_kernel<<<NUM_BLOCK, BLOCK_SIZE>>>(input, bins, num_elements, num_bins);


	  /*************************************************************************/

}


